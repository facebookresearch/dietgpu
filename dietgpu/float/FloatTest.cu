#include "hip/hip_runtime.h"
/**
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <gtest/gtest.h>
#include <cmath>
#include <cstring>
#include <iostream>
#include <random>
#include <vector>

#include "dietgpu/float/GpuFloatCodec.h"
#include "dietgpu/float/GpuFloatUtils.cuh"
#include "dietgpu/utils/StackDeviceMemory.h"

using namespace dietgpu;

uint16_t float32ToBFloat16(float f) {
  // FIXME: does not round to nearest even
  static_assert(sizeof(float) == sizeof(uint32_t), "");
  uint32_t x;
  std::memcpy(&x, &f, sizeof(float));

  x >>= 16;
  return x;
}

uint16_t float32ToFloat16(float f) {
  static_assert(sizeof(float) == sizeof(uint32_t), "");
  uint32_t x;
  std::memcpy(&x, &f, sizeof(float));

  uint32_t u = (x & 0x7fffffff), remainder, shift, lsb, lsb_s1, lsb_m1;
  uint32_t sign, exponent, mantissa;

  // Get rid of +NaN/-NaN case first.
  if (u > 0x7f800000U) {
    return 0x7fffU;
  }

  sign = ((x >> 16) & 0x8000);

  // Get rid of +Inf/-Inf, +0/-0.
  if (u > 0x477fefffU) {
    return sign | 0x7c00U;
  }
  if (u < 0x33000001U) {
    return (sign | 0x0000);
  }

  exponent = ((u >> 23) & 0xff);
  mantissa = (u & 0x7fffff);

  if (exponent > 0x70) {
    shift = 13;
    exponent -= 0x70;
  } else {
    shift = 0x7e - exponent;
    exponent = 0;
    mantissa |= 0x800000;
  }
  lsb = (1 << shift);
  lsb_s1 = (lsb >> 1);
  lsb_m1 = (lsb - 1);

  // Round to nearest even.
  remainder = (mantissa & lsb_m1);
  mantissa >>= shift;
  if (remainder > lsb_s1 || (remainder == lsb_s1 && (mantissa & 0x1))) {
    ++mantissa;
    if (!(mantissa & 0x3ff)) {
      ++exponent;
      mantissa = 0;
    }
  }

  return (sign | (exponent << 10) | mantissa);
}

template <FloatType FT>
struct GenerateFloat;

template <>
struct GenerateFloat<FloatType::kFloat16> {
  static FloatTypeInfo<FloatType::kFloat16>::WordT gen(float v) {
    return float32ToFloat16(v);
  }
};

template <>
struct GenerateFloat<FloatType::kBFloat16> {
  static FloatTypeInfo<FloatType::kBFloat16>::WordT gen(float v) {
    return float32ToBFloat16(v);
  }
};

template <>
struct GenerateFloat<FloatType::kFloat32> {
  static FloatTypeInfo<FloatType::kFloat32>::WordT gen(float v) {
    FloatTypeInfo<FloatType::kFloat32>::WordT out;
    std::memcpy(&out, &v, sizeof(float));
    return out;
  }
};

template <FloatType FT>
std::vector<typename FloatTypeInfo<FT>::WordT> generateFloats(int num) {
  std::mt19937 gen(10 + num);
  std::normal_distribution<float> dist;

  auto out = std::vector<typename FloatTypeInfo<FT>::WordT>(num);
  for (auto& v : out) {
    v = GenerateFloat<FT>::gen(dist(gen));
  }

  return out;
}

template <FloatType FT>
void runBatchPointerTest(
    StackDeviceMemory& res,
    int probBits,
    const std::vector<uint32_t>& batchSizes) {
  using FTI = FloatTypeInfo<FT>;

  // run on a different stream to test stream assignment
  auto stream = CudaStream::makeNonBlocking();

  int numInBatch = batchSizes.size();
  uint32_t totalSize = 0;
  uint32_t maxSize = 0;
  for (auto v : batchSizes) {
    totalSize += v;
    maxSize = std::max(maxSize, v);
  }

  auto maxCompressedSize = getMaxFloatCompressedSize(FT, maxSize);

  auto orig = generateFloats<FT>(totalSize);
  auto orig_dev = res.copyAlloc(stream, orig);

  auto inPtrs = std::vector<const void*>(batchSizes.size());
  {
    uint32_t curOffset = 0;
    for (int i = 0; i < inPtrs.size(); ++i) {
      inPtrs[i] = (const typename FTI::WordT*)orig_dev.data() + curOffset;
      curOffset += batchSizes[i];
    }
  }

  auto enc_dev = res.alloc<uint8_t>(stream, numInBatch * maxCompressedSize);

  auto encPtrs = std::vector<void*>(batchSizes.size());
  {
    for (int i = 0; i < inPtrs.size(); ++i) {
      encPtrs[i] = (uint8_t*)enc_dev.data() + i * maxCompressedSize;
    }
  }

  auto outBatchSize_dev = res.alloc<uint32_t>(stream, numInBatch);

  auto compConfig = FloatCompressConfig(FT, probBits, false);

  floatCompress(
      res,
      compConfig,
      numInBatch,
      inPtrs.data(),
      batchSizes.data(),
      encPtrs.data(),
      outBatchSize_dev.data(),
      stream);

  // Decode data
  auto dec_dev = res.alloc<typename FTI::WordT>(stream, totalSize);

  auto decPtrs = std::vector<void*>(batchSizes.size());
  {
    uint32_t curOffset = 0;
    for (int i = 0; i < inPtrs.size(); ++i) {
      decPtrs[i] = (typename FTI::WordT*)dec_dev.data() + curOffset;
      curOffset += batchSizes[i];
    }
  }

  auto outSuccess_dev = res.alloc<uint8_t>(stream, numInBatch);
  auto outSize_dev = res.alloc<uint32_t>(stream, numInBatch);

  auto decompConfig = FloatDecompressConfig(FT, probBits, false);

  floatDecompress(
      res,
      decompConfig,
      numInBatch,
      (const void**)encPtrs.data(),
      decPtrs.data(),
      batchSizes.data(),
      outSuccess_dev.data(),
      outSize_dev.data(),
      stream);

  auto outSuccess = outSuccess_dev.copyToHost(stream);
  auto outSize = outSize_dev.copyToHost(stream);

  for (int i = 0; i < outSuccess.size(); ++i) {
    EXPECT_TRUE(outSuccess[i]);
    EXPECT_EQ(outSize[i], batchSizes[i]);
  }

  auto dec = dec_dev.copyToHost(stream);

  for (int i = 0; i < orig.size(); ++i) {
    if (orig[i] != dec[i]) {
      printf(
          "mismatch at %d / %d: 0x%08X 0x%08X\n",
          i,
          (int)orig.size(),
          orig[i],
          dec[i]);
      break;
    }
  }

  EXPECT_EQ(orig, dec);
}

void runBatchPointerTest(
    StackDeviceMemory& res,
    FloatType ft,
    int probBits,
    const std::vector<uint32_t>& batchSizes) {
  switch (ft) {
    case FloatType::kFloat16:
      runBatchPointerTest<FloatType::kFloat16>(res, probBits, batchSizes);
      break;
    case FloatType::kBFloat16:
      runBatchPointerTest<FloatType::kBFloat16>(res, probBits, batchSizes);
      break;
    case FloatType::kFloat32:
      runBatchPointerTest<FloatType::kFloat32>(res, probBits, batchSizes);
      break;
    default:
      CHECK(false);
      break;
  }
}

void runBatchPointerTest(
    StackDeviceMemory& res,
    FloatType ft,
    int probBits,
    int numInBatch,
    uint32_t multipleOf = 1) {
  std::mt19937 gen(10 + numInBatch);
  std::uniform_int_distribution<uint32_t> dist(1, 10000);

  auto batchSizes = std::vector<uint32_t>(numInBatch);
  for (auto& v : batchSizes) {
    v = roundUp(dist(gen), multipleOf);
  }

  runBatchPointerTest(res, ft, probBits, batchSizes);
}

TEST(FloatTest, Batch) {
  auto res = makeStackMemory();

  for (auto ft :
       {FloatType::kFloat16, FloatType::kBFloat16, FloatType::kFloat32}) {
    for (auto probBits : {9, 10}) {
      for (auto numInBatch : {1, 3, 16, 23}) {
        runBatchPointerTest(res, ft, probBits, numInBatch);
        // Also test the case where there is uniform 16 byte alignment across
        // all batches
        runBatchPointerTest(res, ft, probBits, numInBatch, 16);
      }
    }
  }
}

TEST(FloatTest, LargeBatch) {
  auto res = makeStackMemory();

  auto batchSizes = std::vector<uint32_t>(256);
  for (auto& v : batchSizes) {
    v = 512 * 1024;
  }

  for (auto ft :
       {FloatType::kFloat16, FloatType::kBFloat16, FloatType::kFloat32}) {
    runBatchPointerTest(res, ft, 10, batchSizes);
  }
}

TEST(FloatTest, BatchSize1) {
  auto res = makeStackMemory();

  for (auto ft :
       {FloatType::kFloat16, FloatType::kBFloat16, FloatType::kFloat32}) {
    for (auto probBits : {9, 10}) {
      runBatchPointerTest(res, ft, probBits, {1});
      runBatchPointerTest(res, ft, probBits, {13, 1});
      runBatchPointerTest(res, ft, probBits, {12345, 1, 8083, 1, 17});
    }
  }
}
